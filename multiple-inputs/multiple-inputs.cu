#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <>
#include <string>
#include <algorithm>

// New threading strategy
// Using block -> num of hidden neurons = blockDim.x, num of inputs = blockDim.y

// Inputs -> weight matrix, biases, x inputs, number of input neurons, number of output neurons
// Outputs -> set values (z values), activation values
// For every output neuron we have to loop over every input neuron -> look at formula
__global__ void multiple_inputs(float *weight_mat, float *biases, float *z_val,
                                float *activation_val, int *shape, int shape_length){
    int idx = threadIdx.x + (blockDim.x * threadIdx.y) + (threadIdx.z * blockDim.x * blockDim.y);
    
    // define offsets for current layer
    // help index into correct layer
    int layer_offset_weights = 0;
    int layer_offset_z = 0;
    int layer_offset_b = 0;
    int layer_offset_activations_input_layer = 0;
    int layer_offset_activations_current_layer = shape[0] * blockDim.y;

    // for multiple layers use variable to define shape of network -> [8,6,4,1]
    // for each layer we calculate the set values and activation values, so loop through every layer
    for(int shape_idx = 0; shape_idx < shape_length; ++shape_idx){
        
        // memory guard -> if(idx < output neurons (same as before)) -> shape[shape_idx + 1] because the number of output neurons
        // is number of neurons in next layer

        // New storage strategy -> z values and activation different, weight and biases the same
        // order of hidden layers the same, but now each layer holds indexes for multiple inputs

        // E.g.
        // z values
        // first hidden layer -> instead of [0,5] (6 neurons), [0,11] (6 neurons x 2 for 2 inputs)
        // second hidden layer -> [12,19] (4 neurons x 2 for 2 inputs)
        // output layer -> [20,21] (1 neuron x 2 for 2 inputs)

        // activation values
        // input layer -> instead of [0,7] (8 neurons), [0,15] (8 neurons x 2 for 2 inputs)
        // first hidden layer -> [16,27] (6 neurons x 2 for 2 inputs)
        // second hidden layer -> [28,35] (4 neurons x 2 for 2 inputs)
        // output layer -> [36,37] (2 neurons x 2 for 2 inputs)

        // 1 thread per input and neuron -> we are using 3 inputs, 6 neurons, so 18 threads
        
        if(idx < shape[shape_idx + 1]){
             
            // each neuron in the layer requires an input
            int num_input_in_layer = shape[shape_idx];
            int layer_size = shape[shape_idx + 1];

            for(int i = 0; i < num_input_in_layer; ++i){

                // weights work by starting at correct offset (layer_offset_weights)
                // then for each output neuron -> number of inputs * idx gives starting point
                // then + i for each input neuron for that output neuron

                // activation values simpler
                // just flat for each layer so as long as offset is correct can index into each neuron
                z_val[layer_offset_z_b + idx] += weight_mat[layer_offset_weights + (num_input_in_layer) * idx + i] * 
                                                activation_val[layer_offset_activations + i];
            }
        
            // w * x + b
            // gives final set values
            z_val[layer_offset_z_b + idx] += biases[layer_offset_z_b + idx];
        
            // activation function
            // sig(w * x + b)
            // + shape[shape_idx] -> write activation values for next layer so input values are not overwritten
            activation_val[layer_offset_activations + shape[shape_idx] + idx] = 1.0 / (1.0 + exp(-z_val[layer_offset_z_b + idx]));
        }
        
        // shift everthing forward as explained in memory layout earlier
        layer_offset_z_b += shape[shape_idx + 1];
        layer_offset_weights += shape[shape_idx] * shape[shape_idx + 1];
        layer_offset_activations += shape[shape_idx];

        // make sure all threads for previous layer complete before moving onto next
        __syncthreads();
    }
}

int main(){

    const int shape_length = 4;
    int shape[shape_length]  = {8, 6, 4, 1};
    const int num_inputs = 3;

    // initialise weights on host
    int num_weights = 0;
    for(int i = 0; i < shape_length - 1; ++i){
        num_weights += shape[i] * shape[i + 1];
    }

    float *h_weights = new float [num_weights] {1.62f, -0.61f, -0.53f, -1.07f, 0.87f, -2.30f, 1.74f, -0.76f, 0.32f, -0.25f, 1.46f, 
                                                -2.06f, -0.32f, -0.38f, 1.13f, -1.10f, -0.17f, -0.88f, 0.04f, 0.58f, -1.10f, 1.14f, 0.90f, 0.50f,
                                                0.90f, -0.68f, -0.12f, -0.94f, -0.27f, 0.53f, -0.69f, -0.40f, -0.69f, -0.85f, -0.67f, -0.01f,
                                                -1.12f, 0.23f, 1.66f, 0.74f, -0.19f, -0.89f, -0.75f, 1.69f, 0.05f, -0.64f, 0.19f, 2.10f, 0.12f,
                                                0.62f, 0.30f, -0.35f, -1.14f, -0.35f, -0.21f, 0.59f, 0.84f, 0.93f, 0.29f, 0.89f, -0.75f, 1.25f, 0.51f,
                                                -0.30f, 0.49f, -0.08f, 1.13f, 1.52f, 2.19f, -1.40f, -1.44f, -0.50f, 0.16f, 0.88f, 0.32f, -2.02f};

    // initialise biases on host
    int num_neurons = 0;
    int num_biases = 0;

    for(int i = 0; i < shape_length; ++i){
        num_neurons += shape[i];
    }

    num_biases = num_neurons - shape[0];

    float *h_biases = new float [num_biases] {-0.31f, 0.83f, 0.23f, 0.76f, -0.22f, -0.20f, 0.19f, 0.41f, 0.20f, 0.12f, -0.67f};

    // initialise activations on host
    // first 8 values are inputs (no need for separate input array anymore since we included it in activations)
    // rest are initialised to 0.0
    int num_activation = num_inputs * num_neurons;
    float *h_activation = new float [num_activation] {0.38f, 0.12f, 1.13f, 1.20f, 0.19f, -0.38f, -0.64f, 0.42f, 0.76f, -0.36f, -0.23f, -0.89f, 
                                                        -0.01f, -0.08f, -0.26f, -0.13f, -0.55f, -0.42f, -0.39f, -0.83f, 0.87f, 0.44f, -0.45f, -0.52f};

    // Initialise z matrix on host
    int num_z = num_biases * num_inputs;
    float *h_z = new float [num_z] {0.0f};

    const size_t size_weights = num_weights * sizeof(float);
    const size_t size_biases = num_biases * sizeof(float);
    const size_t size_activation = num_activation * sizeof(float);
    const size_t size_z = num_z * sizeof(float);
    const size_t size_shape = shape_length * sizeof(int);

    float *d_weights, *d_biases, *d_activation, *d_z;
    int *d_shape;

    hipMalloc(&d_weights, size_weights);
    hipMalloc(&d_biases, size_biases);
    hipMalloc(&d_activation, size_activation);
    hipMalloc(&d_z, size_z);
    hipMalloc(&d_shape, size_shape);

    hipMemcpy(d_weights, h_weights, size_weights, hipMemcpyHostToDevice);
    hipMemcpy(d_biases, h_biases, size_biases, hipMemcpyHostToDevice);
    hipMemcpy(d_activation, h_activation, size_activation, hipMemcpyHostToDevice);
    hipMemcpy(d_z, h_z, size_z, hipMemcpyHostToDevice);
    hipMemcpy(d_shape, shape, size_shape, hipMemcpyHostToDevice);

    // <<<num_blocks, threads_per_block>>>
    // sets width of neural network (ours is 8)
    // finds the largest number of neurons in any layer -> the width
    // dim3 creates 2D block of threads -> (x,y) -> x = num neurons in layer, y = num of inputs being processed
    // e.g. num_threads_x_direction = 8, num_inputs = 2, creates 2D block of 16 threads
    // one thread for each neuron which computes 2 inputs for each neuron
    int num_threads_x_dimensions = *std::max_element(shape + 1, shape + shape_length);
    dim3 thread_block_dimensions(num_threads_x_dimensions, num_inputs);
    multiple_inputs<<<1, thread_block_dimensions>>>(d_weights, d_biases, d_z, d_activation, d_shape, shape_length);

    // Back to host
    hipMemcpy(h_activation, d_activation, size_activation, hipMemcpyDeviceToHost);
    hipMemcpy(h_z, d_z, size_z, hipMemcpyDeviceToHost);

    hipFree(d_weights);
    hipFree(d_biases);
    hipFree(d_activation);
    hipFree(d_z);
    hipFree(d_shape);

    int z_offset = 0;
    for(int shape_idx = 1; shape_idx < shape_length; ++shape_idx){
        std::cout << "Z values: " << shape_idx << ". hidden layer" << std::endl;
        
        for(int i = 0; i < shape[shape_idx]; ++i){
            std::cout << "[";
            for(int j = 0; j < num_inputs; ++j){
                std::cout << h_z[z_offset + j * shape[shape_idx] + i] << ", ";
            }
            std::cout << "]" << std::endl;
        } 
        z_offset += shape[shape_idx] * num_inputs;
    }
    
    int activation_offset = shape[0]; // skip input values
    for(int shape_idx = 1; shape_idx < shape_length; ++shape_idx){
        std::cout << "Activations: " << shape_idx << ". hidden layer" << std::endl;
        for(int i = 0; i < shape[shape_idx]; ++i){
            std::cout << "[";
            for(int j = 0; j < num_inputs; ++j){
                std::cout << h_z[activation_offset + j * shape[shape_idx] + i] << ", ";
            }
            std::cout << "]" << std::endl;
        } 
        activation_offset += shape[shape_idx] * num_inputs;
    }

    getchar();

    return 0;
}